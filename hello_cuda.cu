
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void cuda_hello() {
    const int b = blockIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    printf("Hello World from block %d and thread %d-%d!\n", b, tx, ty);
}
int main() {
    const dim3 block_size(2, 4);
    cuda_hello<<<1, block_size>>>();
    hipDeviceSynchronize();
    return 0;
}
